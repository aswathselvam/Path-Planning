#include "hip/hip_runtime.h"
/**
 * @file space.cpp
 * @author Aswath Muthuselvam (aswath@umd.edu)
 * @brief RRT Logic implementation file.
 * @version 1.0
 * @date 02-15-2022
 * @copyright BSD3 Copyright (c) 2022

   Copyright (c) 2022, Aswath Muthuselvam
   All rights reserved.

   Redistribution and use in source and binary forms, with or without
   modification, are permitted provided that the following conditions are met:

   * Redistributions of source code must retain the above copyright notice, this
     list of conditions and the following disclaimer.

   * Redistributions in binary form must reproduce the above copyright notice,
     this list of conditions and the following disclaimer in the documentation
     and/or other materials provided with the distribution.

   * Neither the name of easyRNG nor the names of its
     contributors may be used to endorse or promote products derived from
     this software without specific prior written permission.
     
   THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
   AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
   IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
   DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
   FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
   DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
   SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
   CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
   OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
   OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include "space.h"
#include <stdlib.h>    
#include <cmath>
#include <iostream>
#include <limits>
#include <typeinfo>
#include <variant>

using std::cout;
using std::endl; 

template class Space<Node, Obstacle>;

Obstacle::Obstacle(){
    this->x=(5)*10;
    this->y=1*10;
    this->r=1*1;
}

Obstacle::Obstacle(int i){
    this->x=(5-i)*10;
    this->y=i*10;
    this->r=i*1;
}

Obstacle3D::Obstacle3D(int i){
    this->x=(5-i)*11;
    this->y=i*11;
    this->z=i*2;
    this->r=i*3;
}

void Node::setup(double x, double y, double z){
    this->x = x;
    this->y = y;
}

void Node3D::setup(double x, double y, double z){
    this->x = x;
    this->y = y;
    this->z = z;
}

template <class NodeDim, class ObstacleDim>
Space<NodeDim, ObstacleDim>::Space() : goalnode(goal){

}

template <class NodeDim, class ObstacleDim>
void Space<NodeDim, ObstacleDim>::init(){
    width=300;
    height=width;
    start.setup(10.0, 10.0, 10.0);
    goal.setup(width-10.2, 50.0, 50.0);

    srand (1);

    N_OF_OBSTACLES=1000; 
    h_obstacles = new ObstacleDim[N_OF_OBSTACLES];
    for(int i = 0; i<N_OF_OBSTACLES ;i++){
        ObstacleDim obstacle(i);
        obstacle.intersected = false;
        obstacle.x=rand() % width;
        obstacle.y=rand() % height;
        obstacle.r=rand() % 3;
        h_obstacles[i] = obstacle;
        this->obstacles.push_back(obstacle);
    }

    hipMalloc(&d_obstacles, N_OF_OBSTACLES*sizeof(ObstacleDim));
	hipMemcpy(d_obstacles, h_obstacles, N_OF_OBSTACLES*sizeof(ObstacleDim), hipMemcpyHostToDevice);
    
}

template <class NodeDim, class ObstacleDim>
NodeDim& Space<NodeDim, ObstacleDim>::addNode(){

    NodeDim* node = new NodeDim{rand() % width + 1.0, rand() % height + 1.0};

    double inf=std::numeric_limits<double>::max();
    NodeDim& nearestnode = getNearestNode(inf,this->start,*node);

    directionComponent(nearestnode, *node);

    bool collision = false;
    bool CUDA=false;
    if(CUDA){
        cudaCheckCollision<<<1, N_OF_OBSTACLES>>>(this, d_obstacles, *node); 
        hipMemcpy(h_obstacles, d_obstacles, N_OF_OBSTACLES*sizeof(ObstacleDim), hipMemcpyDeviceToHost);
        for(int i =0; i< N_OF_OBSTACLES; i++){
            ObstacleDim d = h_obstacles[i];
            if(d.intersected){
                collision=true;
                // cout<<"Collision detected at x: "<<d.x<<" y: "<<d.y<<endl;
            }
        }
    }else{
        collision = checkCollision(*node);
    }
      
    if(collision){
        delete node;
        return addNode();
    }
    addConnection(nearestnode, *node);
    return *node;
}

template <class NodeDim, class ObstacleDim>
__host__ __device__ double Space<NodeDim, ObstacleDim>::L2(Obstacle& obstacle, Node& node){
    return sqrt(pow(obstacle.x-node.x,2) + pow(obstacle.y-node.y,2) );
}

template <class NodeDim, class ObstacleDim>
double Space<NodeDim, ObstacleDim>::L2(Obstacle3D& obstacle, Node3D& node){
    return sqrt(pow(obstacle.x-node.x,2) + pow(obstacle.y-node.y,2) + pow(obstacle.z-node.z,2) );
}

template <class NodeDim, class ObstacleDim>
double Space<NodeDim, ObstacleDim>::L2(Node& n1, Node& n2){
    return sqrt(pow(n1.x-n2.x,2) + pow(n1.y-n2.y,2) );
}

template <class NodeDim, class ObstacleDim>
double Space<NodeDim, ObstacleDim>::L2(Node3D& n1, Node3D& n2){
    return sqrt(pow(n1.x-n2.x,2) + pow(n1.y-n2.y,2) + pow(n1.z-n2.z,2) );
}


template <class NodeDim, class ObstacleDim>
void Space<NodeDim, ObstacleDim>::directionComponent(Node& n1to, Node& n2){
    double mag = L2(n1to, n2);
    double dist= mag > 3 ? 3 : mag;
    n2.x = n1to.x + dist*(n2.x - n1to.x)/mag; 
    n2.y = n1to.y + dist*(n2.y - n1to.y)/mag;
}

template <class NodeDim, class ObstacleDim>
void Space<NodeDim, ObstacleDim>::directionComponent(Node3D& n1to, Node3D& n2){
    double mag = L2(n1to, n2);
    double dist= mag > 3 ? 3 : mag;
    n2.x = n1to.x + dist*(n2.x - n1to.x)/mag; 
    n2.y = n1to.y + dist*(n2.y - n1to.y)/mag;
    n2.z = n1to.z + dist*(n2.z - n1to.z)/mag;
}

template <class NodeDim, class ObstacleDim>
bool Space<NodeDim, ObstacleDim>::checkCollision(NodeDim& node){
    for(ObstacleDim& obstacle : this->obstacles){
        if(L2(obstacle, node) < 2*obstacle.r){
            return true;
        }
    }
    return false;
}

template <class NodeDim, class ObstacleDim>
__global__ void cudaCheckCollision(Space<NodeDim, ObstacleDim>* spacep, Obstacle* d_obstacles, Node node){
    int idx = blockIdx.x + threadIdx.x ;//+ blockDim.x;
    double dist = spacep->L2(d_obstacles[idx], node);
    // printf("idx: %d, dist: %f, radius: %f\n",idx, dist, d_obstacles[idx].r);
    d_obstacles[idx].intersected=false;
    if(dist<2*d_obstacles[idx].r){
        d_obstacles[idx].intersected=true;
    }
}

template <class NodeDim, class ObstacleDim>
void Space<NodeDim, ObstacleDim>::addConnection(NodeDim& a, NodeDim& b){
    a.childNodes.push_back(&b);
    b.parentNode=&a;
}

template <class NodeDim, class ObstacleDim>
NodeDim& Space<NodeDim, ObstacleDim>::getNearestNode(double& min_dist, NodeDim& currentNode, NodeDim& node){
    NodeDim* nearestnode = nullptr;
    if(&currentNode == &start && currentNode.childNodes.size()<1){
        return currentNode;
    }
    for(NodeDim* childNode: currentNode.childNodes){
        float dist = L2(*childNode, node);
        if(dist < min_dist){
            nearestnode = childNode;
            min_dist = dist;
        }
        NodeDim& nearestnodeCandidate = getNearestNode(min_dist, *childNode, node);
        if(&nearestnodeCandidate!=nullptr){
                nearestnode = &nearestnodeCandidate;
        }
    }

    return *nearestnode;
}

template <class NodeDim, class ObstacleDim>
bool Space<NodeDim, ObstacleDim>::solve(){
    NodeDim* node = &addNode();

    if(node!=nullptr){
        if(L2(goal,*node) < 5){
            goalnode = *node;
            return true;
        }
    }
    return false;
}